#include "hip/hip_runtime.h"
﻿// http://stackoverflow.com/questions/3557221/how-do-i-measure-time-in-c
// http://geeksquiz.com/merge-sort/
// http://mc.stanford.edu/cgi-bin/images/3/34/Darve_cme343_cuda_3.pdf


#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <windows.h>

#define N 131072
#define threadSize 1
//#define blockSize N/2


void merge(long arr[], int l, int m, int r)
{
	int i, j, k;
	int n1 = m - l + 1;
	int n2 = r - m;

	/* create temp arrays */
	long L[N/2], R[N/2];

	/* Copy data to temp arrays L[] and R[] */
	for (i = 0; i < n1; i++)
		L[i] = arr[l + i];
	for (j = 0; j < n2; j++)
		R[j] = arr[m + 1 + j];

	/* Merge the temp arrays back into arr[l..r]*/
	i = 0;
	j = 0;
	k = l;
	while (i < n1 && j < n2)
	{
		if (L[i] <= R[j])
		{
			arr[k] = L[i];
			i++;
		}
		else
		{
			arr[k] = R[j];
			j++;
		}
		k++;
	}

	/* Copy the remaining elements of L[], if there are any */
	while (i < n1)
	{
		arr[k] = L[i];
		i++;
		k++;
	}

	/* Copy the remaining elements of R[], if there are any */
	while (j < n2)
	{
		arr[k] = R[j];
		j++;
		k++;
	}
}

void mergeSort(long arr[], int l, int r)
{
	if (l < r)
	{
		int m = l + (r - l) / 2; //Same as (l+r)/2, but avoids overflow for large l and h
		mergeSort(arr, l, m);
		mergeSort(arr, m + 1, r);
		merge(arr, l, m, r);
	}
}


void printArray(long A[], int size)
{
	int i;
	for (i = 0; i < size; i++)
		printf("%d ", A[i]);
	printf("\n");
}

__global__ void gpu_MergeSort(long* source, long *dest, long size) {
	long index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		long start = index * size;

		long middle = start + size / 2;

		long end = start + size;
		if (end > N)
		{
			end = N;
		}

		//printf("start: %d - Middle: %d - End: %d\n", start, middle, end);

		long i = start, j = middle;

		long k = start;
		while (i < middle && j < end) {
			if (source[i] <= source[j]) {
				dest[k] = source[i];
				i++;
			}
			else {
				dest[k] = source[j];
				j++;
			}
			k++;
		}
		while (i < middle) {
			dest[k] = source[i];
			k++;
			i++;
		}
		while (j < end) {
			dest[k] = source[j];
			k++;
			j++;
		}
	}
	__syncthreads();
}


int main()
{
	DWORD dwStartTime, dwElapsed;

    long a[N], b[N], *d_A, *d_B;
	
	for (size_t i = 0; i < N; i++)
	{
		a[i] = N - i;
		b[i] = N - i;
	}

	int arr_size = sizeof(b) / sizeof(b[0]);

	//printf("Given array is \n");
	//printArray(b, arr_size);
	dwStartTime = GetTickCount();
	// MERGE SORT WITH CPU
	mergeSort(b, 0, arr_size - 1);

	dwElapsed = GetTickCount() - dwStartTime;
	//printf("\nSorted array is \n");
	//printArray(b, arr_size);
	printf("Calculations with CPU took %d.%3d seconds to complete\n", dwElapsed / 1000, dwElapsed - dwElapsed / 1000);

	int size = N * sizeof(long);

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
	
	long blockSize = 0;
	
	dwStartTime = GetTickCount();
	// MERGE SORT WITH GPU
	for (size_t i = 2; i <= N; i=i*2)
	{
		blockSize = N / (threadSize * i);
		//printf("block: % d - thd: %d - i: %d\n", blockSize, threadSize, i);
		gpu_MergeSort <<<blockSize, threadSize>> >(d_A, d_B, i);
		hipDeviceSynchronize();
		//hipMemcpy(a, d_B, size, hipMemcpyDeviceToHost);
		//printArray(a, arr_size);
		// Swap source with destination array
		long *temp = d_A;
		d_A = d_B;
		d_B = temp;
	}
	dwElapsed = GetTickCount() - dwStartTime;
	//printArray(a, arr_size);

	printf("Calculations with GPU took %d.%3d seconds to complete\n", dwElapsed / 1000, dwElapsed - dwElapsed / 1000);
    return 0;
}
